#include "hip/hip_runtime.h"
/*
 * Matrix is a PHP extension. It can do parallel computing base on CUDA.
 *
 * GitHub: https://github.com/BourneSuper/matrix
 *
 * Author: Bourne Wong <cb44606@gmail.com>
 *
 * */


#include <stdio.h>

// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>

#include "math.cuh"



int getMaxThreadsPerMultiProcessor(  deviceContextStruct * deviceContextStructP ){
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties( &deviceProp, deviceContextStructP->deviceId );
    
    return deviceProp.maxThreadsPerMultiProcessor;
}



//arrayAdd()
__global__ void arrayAddKernel( double *deviceA, double alpha, int elementNum ){
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if( i < elementNum) {
        deviceA[i] = deviceA[i] + alpha;
    }
}

void arrayAdd( deviceContextStruct * deviceContextStructP, double * hostAP, int elementNum, double alpha ){
    int sizeA = elementNum * sizeof(double);
    
    //
    double * deviceA;
    hipMalloc( (void **) &deviceA, sizeA );

    //
    hipMemcpy( deviceA, hostAP, sizeA, hipMemcpyHostToDevice );
    
    //
    int threadsPerBlock = getMaxThreadsPerMultiProcessor( deviceContextStructP );
    int blocksPerGrid = ( elementNum + threadsPerBlock - 1 ) / threadsPerBlock;
    
    arrayAddKernel<<< blocksPerGrid, threadsPerBlock >>>( deviceA, alpha, elementNum );
    
    //
    hipMemcpy( hostAP, deviceA, sizeA, hipMemcpyDeviceToHost );
    
    
    hipFree(deviceA);
    
}


//subtractArray()
__global__ void subtractArrayKernel(  double alpha, double *deviceA, int elementNum ){
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if( i < elementNum) {
        deviceA[i] = alpha - deviceA[i];
    }
}

void subtractArray( deviceContextStruct * deviceContextStructP,  double alpha, double * hostAP, int elementNum ){
    int sizeA = elementNum * sizeof(double);
    
    //
    double * deviceA;
    hipMalloc( (void **) &deviceA, sizeA );

    //
    hipMemcpy( deviceA, hostAP, sizeA, hipMemcpyHostToDevice );
    
    //
    int threadsPerBlock = getMaxThreadsPerMultiProcessor( deviceContextStructP );
    int blocksPerGrid = ( elementNum + threadsPerBlock - 1 ) / threadsPerBlock;
    
    subtractArrayKernel<<< blocksPerGrid, threadsPerBlock >>>( alpha, deviceA, elementNum );
    
    //
    hipMemcpy( hostAP, deviceA, sizeA, hipMemcpyDeviceToHost );
    
    
    hipFree(deviceA);
    
}


//arrayMultiply()
__global__ void arrayMultiplyKernel( double *deviceA, double alpha, int elementNum ){
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if( i < elementNum) {
        deviceA[i] = deviceA[i] * alpha;
    }
}

void arrayMultiply( deviceContextStruct * deviceContextStructP, double * hostAP, int elementNum, double alpha ){
    int sizeA = elementNum * sizeof(double);
    
    //
    double * deviceA;
    hipMalloc( (void **) &deviceA, sizeA );

    //
    hipMemcpy( deviceA, hostAP, sizeA, hipMemcpyHostToDevice );
    
    //
    int threadsPerBlock = getMaxThreadsPerMultiProcessor( deviceContextStructP );
    int blocksPerGrid = ( elementNum + threadsPerBlock - 1 ) / threadsPerBlock;
    
    arrayMultiplyKernel<<< blocksPerGrid, threadsPerBlock >>>( deviceA, alpha, elementNum );
    
    //
    hipMemcpy( hostAP, deviceA, sizeA, hipMemcpyDeviceToHost );
    
    
    hipFree(deviceA);
    
}


//divideArray()
__global__ void divideArrayKernel(  double alpha, double *deviceA, int elementNum ){
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if( i < elementNum) {
        deviceA[i] = alpha / deviceA[i];
    }
}

void divideArray( deviceContextStruct * deviceContextStructP,  double alpha, double * hostAP, int elementNum ){
    int sizeA = elementNum * sizeof(double);
    
    //
    double * deviceA;
    hipMalloc( (void **) &deviceA, sizeA );

    //
    hipMemcpy( deviceA, hostAP, sizeA, hipMemcpyHostToDevice );
    
    //
    int threadsPerBlock = getMaxThreadsPerMultiProcessor( deviceContextStructP );
    int blocksPerGrid = ( elementNum + threadsPerBlock - 1 ) / threadsPerBlock;
    
    divideArrayKernel<<< blocksPerGrid, threadsPerBlock >>>( alpha, deviceA, elementNum );
    
    //
    hipMemcpy( hostAP, deviceA, sizeA, hipMemcpyDeviceToHost );
    
    
    hipFree(deviceA);
    
}


//arrayPower()
__global__ void arrayPowerKernel( double *deviceA, double alpha, int elementNum ){
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if( i < elementNum) {
        deviceA[i] =  pow( deviceA[i], alpha );
    }
}

void arrayPower( deviceContextStruct * deviceContextStructP, double * hostAP, int elementNum, double alpha ){
    int sizeA = elementNum * sizeof(double);
    
    //
    double * deviceA;
    hipMalloc( (void **) &deviceA, sizeA );

    //
    hipMemcpy( deviceA, hostAP, sizeA, hipMemcpyHostToDevice );
    
    //
    int threadsPerBlock = getMaxThreadsPerMultiProcessor( deviceContextStructP );
    int blocksPerGrid = ( elementNum + threadsPerBlock - 1 ) / threadsPerBlock;
    
    arrayPowerKernel<<< blocksPerGrid, threadsPerBlock >>>( deviceA, alpha, elementNum );
    
    //
    hipMemcpy( hostAP, deviceA, sizeA, hipMemcpyDeviceToHost );
    
    
    hipFree(deviceA);
    
}


//hadamardProduct()
__global__ void hadamardProductKernel( double * deviceA, double * deviceB, int elementNum ){
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    
    if( i < elementNum ){
        deviceA[i] = deviceA[i] * deviceB[i];
    }
}

void hadamardProduct( deviceContextStruct * deviceContextStructP, double * hostAP, double * hostBP, int elementNum ){
    int sizeA = elementNum * sizeof(double);
    
    //
    double * deviceA, * deviceB;
    hipMalloc( (void **) &deviceA, sizeA );
    hipMalloc( (void **) &deviceB, sizeA );

    //
    hipMemcpy( deviceA, hostAP, sizeA, hipMemcpyHostToDevice );
    hipMemcpy( deviceB, hostBP, sizeA, hipMemcpyHostToDevice );
    
    //
    int threadsPerBlock = getMaxThreadsPerMultiProcessor( deviceContextStructP );
    int blocksPerGrid = ( elementNum + threadsPerBlock - 1 ) / threadsPerBlock;
    
    hadamardProductKernel<<< blocksPerGrid, threadsPerBlock >>>( deviceA, deviceB, elementNum );
    
    //
    hipMemcpy( hostAP, deviceA, sizeA, hipMemcpyDeviceToHost );
    
    
    hipFree(deviceA);
    
}






/*
 * Matrix is a PHP extension. It can do parallel computing base on CUDA.
 *
 * GitHub: https://github.com/BourneSuper/matrix
 *
 * Author: Bourne Wong <cb44606@gmail.com>
 *
 * */


